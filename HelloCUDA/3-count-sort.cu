#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "strhelper.h"
#include "hip/hip_runtime.h"
#include ""
using namespace std;
int maxn, * src, * sorted, err_detecter;
__global__ void numAdd(int* a, int* f)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	//atomicAdd(&f[i], 1);
}
extern "C" int sort_3()
{
	printf("Choose one way to test: \n");
	printf("\t1. Type in all data.\n");
	printf("\t2. Use random data produced by computer.\n");
	printf("Your selection: ");
	string in;
	getline(cin, in);
	int chosen = int_parse(&in);
	switch (chosen)
	{
		case 1:
			printf("Type the max num of the data: ");
			err_detecter = scanf("%d", &maxn);
			src = new int[maxn], sorted = new int[maxn];
			printf("Array init ...\nType your data one by one separated by space:\n");
			for (int i = 0; i < maxn; ++i) err_detecter = scanf("%d", &src[i]);
			printf("Data inputing ended !\n");
			break;
		case 2:
			printf("Starting generate random data ...\n");
			maxn = 10000000;
			src = new int[maxn], sorted = new int[maxn];
			memset(sorted, 0, sizeof(int) * maxn);
			srand(time(0));
			for (int i = 0; i < maxn; ++i) src[i] = rand() % 100000;
			printf("Random data generated !\n");
			break;
		default: printf("No this selection.\n"); break;
	}
	int* d_src, * d_rst;
	#pragma region CUDA
	hipMalloc((void**)&d_src, sizeof(int) * maxn);
	hipMalloc((void**)&d_rst, sizeof(int) * maxn);
	hipMemcpy(d_src, src, sizeof(int) * maxn, hipMemcpyHostToDevice);
	hipMemcpy(d_rst, sorted, sizeof(int) * maxn, hipMemcpyHostToDevice);
	numAdd << <1000, 10000 >> > (d_src, d_rst);
	#pragma endregion
	delete[] src, sorted;
	return 0;
}