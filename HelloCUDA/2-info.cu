#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
extern "C" int info_2()
{
    int device_Count = 0;
    hipGetDeviceCount(&device_Count);
    if (device_Count == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n\n", device_Count);
        for (int i = 0; i < device_Count; i++)
        {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
            int driverVersion = 0, runtimeVersion = 0;
            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            printf("Device %d: \"%s\"\n", i + 1, deviceProp.name);
            printf("\tCUDA Driver Version / Runtime Version: %d.%d / %d.%d\n",
                driverVersion / 1000, (driverVersion % 100) / 10,
                runtimeVersion / 1000, (runtimeVersion % 100) / 10);
            printf("\tTotal amount of global memory: %lld MB (%lld Bytes)\n",
                deviceProp.totalGlobalMem / 1024 / 1024, deviceProp.totalGlobalMem);
            printf("\tCUDA Capability Major/Minor version number: %d.%d\n",
                deviceProp.major, deviceProp.minor);
            printf("\tGPU Clock rate: %.0f MHz (%0.2f GHz)\n",
                deviceProp.clockRate * 1.0 * 1e-3f, deviceProp.clockRate * 1.0 * 1e-6f);
            printf("\tMemory Clock rate: %d Mhz\n", deviceProp.memoryClockRate / 1024);
            printf("\tMemory Bus Width: %d-bit\n", deviceProp.memoryBusWidth);
            if (deviceProp.l2CacheSize)
                printf("\tL2 Cache Size: %d bytes\n", deviceProp.l2CacheSize);
            printf("\tMax Texture Dimension Size (x,y,z): 1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
                deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
                deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
                deviceProp.maxTexture3D[2]);
            printf("\tMax Layered 1D Texture Size, (num) layers: 1D=(%d) x %d\n",
                deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
            printf("\tMax Layered 2D Texture Size, (num) layers: 2D=(%d,%d) x %d\n",
                deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
                deviceProp.maxTexture2DLayered[2]);
            printf("\tTotal amount of constant memory: %lld bytes\n", deviceProp.totalConstMem);
            printf("\tTotal amount of shared memory per block: %lld Bytes\n",
                deviceProp.sharedMemPerBlock / 1024);
            printf("\tTotal number of registers available per block: %d\n",
                deviceProp.regsPerBlock);
            printf("\tWarp size: %d\n", deviceProp.warpSize);
            printf("\tMaximum number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
            printf("\tMaximum number of threads per multiprocessor: %d\n",
                deviceProp.maxThreadsPerMultiProcessor);
            printf("\tMaximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
            printf("\tMaximum sizes of each dimension of a block:  %d x %d x %d\n",
                deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
                deviceProp.maxThreadsDim[2]);
            printf("\tMaximum sizes of each dimension of a grid:  %d x %d x %d\n",
                deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
            printf("\tMaximum memory pitch %lld bytes\n", deviceProp.memPitch);
        }
    }
    return 0;
}