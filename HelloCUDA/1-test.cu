#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include ""
using namespace std;
const int maxn = 1000000; // Ԫ��ֵ���� - һ����
const int maxl = 1000000; // ���鳤 - һ����
int* h_a, * h_b, * h_sum, * h_cpusum, tot = 0; // ����ָ��
clock_t pro_start, pro_end, sum_start, sum_end; // CPU ʱ�Ӽ���
hipEvent_t e_start, e_stop; // CUDA ���ܲ��Ե�
__global__ void numAdd(int* a, int* b, int* sum) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    sum[i] = a[i] + b[i];
}
extern "C" int test_1() {
    srand(time(0)); // �趨���������
    h_a = new int[maxl], h_b = new int[maxl], h_sum = new int[maxl], h_cpusum = new int[maxl]; // ��̬�����ڴ�
    pro_start = clock();
    for (int i = 0; i < maxl; ++i) {
        h_a[i] = rand() % maxn; h_b[i] = rand() % maxn; // ��ʼ���������
    }
    pro_end = clock();
    // ��� ��������� ��ʱ ������ CUDA ���޷����������ĳ�ʼ�����첻���жԱȣ�
    printf("produce %d random number to two array use time : %0.3f ms\n", maxl, double(pro_end - pro_start) / CLOCKS_PER_SEC * 1000);
    system("pause");
    int* d_a, * d_b, * d_sum;
    hipEventCreate(&e_start); hipEventCreate(&e_stop); // ���� CUDA �¼�
    hipEventRecord(e_start, 0); // ��¼ CUDA �¼�
    // ���Դ��Ϸ��䵥Ԫ
    hipMalloc((void**)&d_a, sizeof(int) * maxl);
    hipMalloc((void**)&d_b, sizeof(int) * maxl);
    hipMalloc((void**)&d_sum, sizeof(int) * maxl);
    /*printf("memory allocated!\n");*/
    // �����ڴ����ݵ��Դ�
    hipMemcpy(d_a, h_a, sizeof(int) * maxl, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * maxl, hipMemcpyHostToDevice);
    /*printf("copy finished!\n");*/
    numAdd << <1000, 1000 >> > (d_a, d_b, d_sum); // ���� Kernel �������������ִ�д���
    hipDeviceSynchronize(); // ͬ��GPU�ϵ������̣߳��ȴ������߳̽������ټ���
    /*printf("sum finished!\ncopy backing...\n");*/
    hipMemcpy(h_sum, d_sum, sizeof(int) * maxl, hipMemcpyDeviceToHost); // ���Դ濽��������ݵ��ڴ�
    /*printf("copy backed!\n");*/
    hipFree(d_a); hipFree(d_b); hipFree(d_sum); // �ͷ�GPU�Ϸ�����Դ�
    /*printf("device RAM released!\n");*/
    hipEventRecord(e_stop, 0); hipEventSynchronize(e_stop); // ��¼��ʱ
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, e_start, e_stop);
    printf("CUDA sum use time : %0.3f ms\n", elapsedTime);
    sum_start = clock();
    for (int i = 0; i < maxl; ++i) h_cpusum[i] = h_a[i] + h_b[i]; // CPU ���мӷ�����
    sum_end = clock();
    printf("CPU  sum use time : %0.3f ms\n", double(pro_end - pro_start) / CLOCKS_PER_SEC * 1000);
    for (int i = 0; i < maxl; ++i) if (h_cpusum[i] != h_sum[i]) ++tot; // ���� CPU ������ �� GPU �������Ƿ�һ�£�ͳ�Ʋ�һ�¸���
    printf("error sum num : %d\n", tot);
    system("pause");
    delete h_a; delete h_b; delete h_sum; // �ͷ��ڴ�
    return 0;
}